#include "hip/hip_runtime.h"
// headers in STL
#include <stdio.h>

// headers in local files
#include "common.h"
#include "boolvfe.h"

__global__ void make_pillar_histo_kernel(
    const float* dev_points, 
    float* pillar_count_histo, const int num_points,
    const int grid_x_size, const int grid_y_size, const int grid_z_size, 
    const float min_x_range, const float min_y_range, const float min_z_range, 
    const float pillar_x_size, const float pillar_y_size, const float pillar_z_size,
    const int input_point_feature) {
  int th_i = blockIdx.x * blockDim.x +  threadIdx.x ;
  if (th_i >= num_points) {
    return;
  }
  int x_coor = floor((dev_points[th_i * input_point_feature + 0] - min_x_range) / pillar_x_size);
  int y_coor = floor((dev_points[th_i * input_point_feature + 1] - min_y_range) / pillar_y_size);
  int z_coor = floor((dev_points[th_i * input_point_feature + 2] - min_z_range) / pillar_z_size);

  if (x_coor >= 0 && x_coor < grid_x_size && y_coor >= 0 &&
      y_coor < grid_y_size && z_coor >= 0 && z_coor < grid_z_size) {
    pillar_count_histo[z_coor * grid_x_size * grid_y_size + y_coor * grid_x_size + x_coor] = 1;
  }
}

BoolVFECuda::BoolVFECuda(
    const int num_threads, 
    const int input_point_feature,
    const int num_point_feature,
    const int grid_x_size, const int grid_y_size,
    const int grid_z_size, const float pillar_x_size, const float pillar_y_size,
    const float pillar_z_size, const float min_x_range, const float min_y_range,
    const float min_z_range)
    : num_threads_(num_threads),
      input_point_feature_(input_point_feature),
      num_point_feature_(num_point_feature),
      grid_x_size_(grid_x_size),
      grid_y_size_(grid_y_size),
      grid_z_size_(grid_z_size),
      pillar_x_size_(pillar_x_size),
      pillar_y_size_(pillar_y_size),
      pillar_z_size_(pillar_z_size),
      min_x_range_(min_x_range),
      min_y_range_(min_y_range),
      min_z_range_(min_z_range) {
    
  }

BoolVFECuda::~BoolVFECuda() {
  }

void BoolVFECuda::DoBoolVFECuda(
    const float* dev_points, const int in_num_points, 
    float* dev_pillar_count_histo) {
    // initialize paraments

    GPU_CHECK(hipMemset(dev_pillar_count_histo, 0 , grid_y_size_ * grid_x_size_ * grid_z_size_ * sizeof(float)));
    int num_block = DIVUP(in_num_points , num_threads_);

    // std::cout << grid_x_size_ << " " << grid_y_size_ << " " << grid_z_size_ << " " << in_num_points << " " << input_point_feature_ << " "
    // << num_block  << " " << num_threads_ << std::endl;
    make_pillar_histo_kernel<<<num_block , num_threads_>>>(
        dev_points, dev_pillar_count_histo, in_num_points, 
        grid_x_size_, grid_y_size_, grid_z_size_, 
        min_x_range_, min_y_range_, min_z_range_, 
        pillar_x_size_, pillar_y_size_, pillar_z_size_, 
        input_point_feature_);

    // std::cout << grid_x_size_ << " " << grid_y_size_ << " " << grid_z_size_ << " " << 
    //     min_x_range_ << " " << min_y_range_ << " " << min_z_range_ << " " << 
    //     pillar_x_size_ << " " << pillar_y_size_ << " " << pillar_z_size_ << "\n";
    // float * host_point = new float[in_num_points](); 
    // GPU_CHECK(hipMemcpy(host_point, dev_points, in_num_points * input_point_feature_ * sizeof(float), hipMemcpyDeviceToHost));
    // for (int i = 0; i < in_num_points; i++){
    //   // int z_coor = floor((host_point[i * input_point_feature_ + 2] - min_z_range_) / pillar_z_size_);
    //   // if (z_coor == 38) 
    //   //    cout << i << " " << host_point[i * input_point_feature_ + 2] << "\n";
    // }
    // float * temp = new float[grid_y_size_ * grid_x_size_ * grid_z_size_](); 
    // GPU_CHECK(hipMemcpy(temp, dev_pillar_count_histo, grid_y_size_ * grid_x_size_ * grid_z_size_ * sizeof(float), hipMemcpyDeviceToHost));
    // int sum = 0;
    // for (int i = 0; i < grid_z_size_; i++){
    //     int cnt = 0;
    //     for (int j = 0; j < grid_y_size_ * grid_x_size_; j++){
    //         cnt += temp[i * grid_y_size_ * grid_x_size_ + j];
    //     }
    //     sum += cnt;
    //     std::cout << i << " " << cnt << "\n";
    // }
    // // std::cout << "total: " << sum << "\n";
    // for (int i = 0; i < 100; i++){
    //     for (int j = 80; j < 180; j++){
    //         cout << temp[0 * grid_y_size_ * grid_x_size_ + j * grid_x_size_ + i] ;
    //     }
    //     std::cout << "\n";
    // }
}